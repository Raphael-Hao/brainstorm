#include "../balancing.cuh"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main(int argc, char* args[]) {
    int n_worker = atoi(args[1]);
    int n_expert = atoi(args[2]);
    int cap_v = atoi(args[3]);
    int tot_expert = n_worker * n_expert;

    long* lec = new long[tot_expert];
    for (int i = 0; i < tot_expert; ++i) {
        lec[i] = i;
    }
    long* g_lec;
    hipMalloc(&g_lec, sizeof(long) * tot_expert);
    hipMemcpy(g_lec, lec, sizeof(long) * tot_expert, hipMemcpyHostToDevice);

    int* cap = new int[n_expert];
    for (int i = 0; i < n_expert; ++i) {
        cap[i] = cap_v;
    }
    int* g_cap;
    hipMalloc(&g_cap, sizeof(int) * n_expert);
    hipMemcpy(g_cap, cap, sizeof(int) * n_expert, hipMemcpyHostToDevice);

    long* eca = new long[tot_expert];
    long* g_eca;
    hipMalloc(&g_eca, sizeof(long) * tot_expert);

    auto smgr = getCudaStreamManager(0);
    fmoe_cuda_limit_by_capacity_impl(g_lec, g_cap, g_eca, n_expert, n_worker, smgr);

    hipMemcpy(cap, g_cap, sizeof(int) * n_expert, hipMemcpyDeviceToHost);
    hipMemcpy(eca, g_eca, sizeof(long) * tot_expert, hipMemcpyDeviceToHost);

    printf("%d\n", cap[0]);
    for (int i = 0; i < tot_expert; ++i) {
        printf("%ld %ld\n", lec[i], eca[i]);
    }
}
