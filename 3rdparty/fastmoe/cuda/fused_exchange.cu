#include "moe_cuda_kernel.h"

#include <cstdio>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_stream_manager.h"
#include "cublas_wrapper.h"

#ifdef FMOE_USE_NCCL
#include <nccl.h>

template<typename scalar_t>
void moe_cuda_global_fused_forward_impl(
        const scalar_t* input_buf,
        const scalar_t* weight,
        scalar_t* global_input_buf,
        scalar_t* global_output_buf,
        scalar_t* output_buf,
        const long* local_expert_count, 
        const long* global_expert_count, 
        long in_feat, long out_feat, 
        long num_expert, long world_size,
        CudaStreamManager* smgr) {

    int ptr = 0;
    int send_ptr = 0;
    int recv_ptr = 0;

    int *expert_ptr = new int[num_expert * world_size];
    expert_ptr[0] = 0;
    for (int i = 1; i < num_expert * world_size; ++i) {
        expert_ptr[i] = expert_ptr[i - 1] + local_expert_count[i - 1];
    }

    scalar_t alpha = 1, beta = 0; 

    for (int i = 0; i < num_expert; ++i) {
        int expert_count = 0;
        NCCL_SAFE_CALL(ncclGroupStart());
        for (int j = 0; j < world_size; ++j) {
            int idx = i + j * num_expert;
            if (local_expert_count[idx]) {
                NCCL_SAFE_CALL(ncclSend(
                        input_buf + expert_ptr[idx] * in_feat, 
                        local_expert_count[idx] * in_feat * sizeof(scalar_t),
                        ncclChar, 
                        j,
                        smgr->ncclcomm,
                        smgr->stream(i)));
            }
            if (global_expert_count[idx]) {
                NCCL_SAFE_CALL(ncclRecv(
                        global_input_buf + recv_ptr * in_feat,
                        global_expert_count[idx] * in_feat * sizeof(scalar_t),
                        ncclChar,
                        j,
                        smgr->ncclcomm,
                        smgr->stream(i)));
                recv_ptr += global_expert_count[idx];
                expert_count += global_expert_count[idx];
            }
        }
        NCCL_SAFE_CALL(ncclGroupEnd());

        checkCudaErrors(cublasXgemm(
                smgr->handle(i),
                HIPBLAS_OP_T,
                HIPBLAS_OP_N,
                out_feat, expert_count, in_feat,
                &alpha,
                weight + i * in_feat * out_feat, in_feat,
                global_input_buf + ptr * in_feat, in_feat,
                &beta,
                global_output_buf + out_feat * ptr, out_feat
                ));

        ptr += expert_count;

        NCCL_SAFE_CALL(ncclGroupStart());
        for (int j = 0; j < world_size; ++j) {
            int idx = i + j * num_expert;
            if (global_expert_count[idx]) {
                NCCL_SAFE_CALL(ncclSend(
                        global_output_buf + send_ptr * out_feat,
                        global_expert_count[idx] * out_feat * sizeof(scalar_t),
                        ncclChar,
                        j,
                        smgr->ncclcomm,
                        smgr->stream(i)));
                send_ptr += global_expert_count[idx];
            }
            if (local_expert_count[idx]) {
                NCCL_SAFE_CALL(ncclRecv(
                        output_buf + expert_ptr[idx] * out_feat, 
                        local_expert_count[idx] * out_feat * sizeof(scalar_t),
                        ncclChar, 
                        j,
                        smgr->ncclcomm,
                        smgr->stream(i)));
            }
        }
        NCCL_SAFE_CALL(ncclGroupEnd());
    }
    delete [] expert_ptr;
    smgr->sync(num_expert);
}

std::vector<torch::Tensor> moe_cuda_global_fused_forward(
        torch::Tensor input_buf,
        torch::Tensor weight,
        torch::Tensor local_expert_count,
        torch::Tensor global_expert_count,
        long global_batch_size, long local_batch_size, long n_workers) {
    const auto num_expert = local_expert_count.size(0) / n_workers;
    const auto out_feat = weight.size(1);
    const auto in_feat = weight.size(2);

    auto smgr = getCudaStreamManager(input_buf.device().index());

    auto global_input_buf = input_buf.new_empty({global_batch_size, in_feat});
    auto global_output_buf = input_buf.new_empty({global_batch_size, out_feat});
    auto output_buf = input_buf.new_empty({local_batch_size, out_feat});
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_buf.scalar_type(), 
            "moe_cuda_global_fused_forward", ([&] {
        moe_cuda_global_fused_forward_impl(
            input_buf.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            global_input_buf.data_ptr<scalar_t>(),
            global_output_buf.data_ptr<scalar_t>(),
            output_buf.data_ptr<scalar_t>(),
            local_expert_count.data_ptr<long>(),
            global_expert_count.data_ptr<long>(),
            in_feat, out_feat, num_expert, n_workers,
            smgr);
    }));
    return {output_buf, global_input_buf};
}

#endif

