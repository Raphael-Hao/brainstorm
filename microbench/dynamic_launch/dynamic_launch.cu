#include "hip/hip_runtime.h"
#include "op.cuh"

int main(int argc, char **argv) {
  // create stream
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  // set stream persistent L2 cache attributes
  hipLaunchAttributeID stream_attr_id = hipLaunchAttributeAccessPolicyWindow;
  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
  size_t persist_size =
      min(int(prop.l2CacheSize * .1), prop.persistingL2CacheMaxSize);
  hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, persist_size);
  hipLaunchAttributeValue stream_attr;
  int *schedule_flag;
  CUDA_CHECK(hipMalloc(&schedule_flag, 16 * sizeof(int)));
  stream_attr.accessPolicyWindow.base_ptr = schedule_flag;
  stream_attr.accessPolicyWindow.num_bytes = 16 * sizeof(int);
  stream_attr.accessPolicyWindow.hitRatio = 1.0;
  stream_attr.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
  stream_attr.accessPolicyWindow.missProp = hipAccessPropertyStreaming;
  CUDA_CHECK(hipStreamSetAttribute(stream, stream_attr_id, &stream_attr));

  float *d_A, *d_B, *d_C;
  int size_A = 1024 * 1024, size_B = size_A, size_C = size_A;
  CUDA_CHECK(hipMalloc(&d_A, size_A * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_B, size_B * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_C, size_C * sizeof(float)));

  // create grid and block
  int block_size = 32;
  dim3 threads = dim3(block_size, block_size);
  dim3 grid(size_A / (threads.x * threads.y), 1);

  // set scheduler flag
  CUDA_CHECK(hipMemsetAsync(schedule_flag, 0, 16 * sizeof(int), stream));

  // launch candidate kernels
  kernel_add<0>
      <<<grid, threads, 0, stream>>>(d_A, d_B, d_C, size_A, schedule_flag);
  kernel_add<1>
      <<<grid, threads, 0, stream>>>(d_A, d_B, d_C, size_A, schedule_flag);
  kernel_add<2>
      <<<grid, threads, 0, stream>>>(d_A, d_B, d_C, size_A, schedule_flag);
  kernel_add<3>
      <<<grid, threads, 0, stream>>>(d_A, d_B, d_C, size_A, schedule_flag);
  CUDA_CHECK(hipStreamSynchronize(stream));

}