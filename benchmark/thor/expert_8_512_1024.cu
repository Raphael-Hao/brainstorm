
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) default_function_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_batch_matmul_NT) {
  float T_batch_matmul_NT_local[8];
  __shared__ float placeholder_d_shared[128];
  __shared__ float placeholder_shared[256];
  T_batch_matmul_NT_local[(0)] = 0.000000e+00f;
  T_batch_matmul_NT_local[(1)] = 0.000000e+00f;
  T_batch_matmul_NT_local[(2)] = 0.000000e+00f;
  T_batch_matmul_NT_local[(3)] = 0.000000e+00f;
  T_batch_matmul_NT_local[(4)] = 0.000000e+00f;
  T_batch_matmul_NT_local[(5)] = 0.000000e+00f;
  T_batch_matmul_NT_local[(6)] = 0.000000e+00f;
  T_batch_matmul_NT_local[(7)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    ((float4*)(placeholder_d_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder + ((((((((int)blockIdx.x) >> 6) * 4096) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
    ((float4*)(placeholder_d_shared + (((((int)threadIdx.x) * 4) + 64))))[0] = ((float4*)(placeholder + (((((((((int)blockIdx.x) >> 6) * 4096) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 2048))))[0];
    ((float4*)(placeholder_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder1 + (((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 64))))[0] = ((float4*)(placeholder1 + ((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 2048))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 128))))[0] = ((float4*)(placeholder1 + ((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 4096))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 192))))[0] = ((float4*)(placeholder1 + ((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 6144))))[0];
    __syncthreads();
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 16))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(16)] * placeholder_shared[((((int)threadIdx.x) * 16))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(32)] * placeholder_shared[((((int)threadIdx.x) * 16))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(48)] * placeholder_shared[((((int)threadIdx.x) * 16))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(64)] * placeholder_shared[((((int)threadIdx.x) * 16))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(80)] * placeholder_shared[((((int)threadIdx.x) * 16))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(96)] * placeholder_shared[((((int)threadIdx.x) * 16))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(112)] * placeholder_shared[((((int)threadIdx.x) * 16))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(17)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(33)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(49)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(65)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(81)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(97)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(113)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(18)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(34)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(50)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(66)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(82)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(98)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(114)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(19)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(35)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(51)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(67)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(83)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(99)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(115)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(20)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(36)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(52)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(68)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(84)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(100)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(116)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(21)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(37)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(53)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(69)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(85)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(101)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(117)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(22)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(38)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(54)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(70)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(86)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(102)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(118)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(23)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(39)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(55)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(71)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(87)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(103)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(119)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(24)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(40)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(56)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(72)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(88)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(104)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(120)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(25)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(41)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(57)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(73)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(89)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(105)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(121)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(26)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(42)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(58)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(74)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(90)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(106)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(122)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(27)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(43)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(59)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(75)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(91)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(107)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(123)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(28)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(44)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(60)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(76)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(92)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(108)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(124)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(29)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(45)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(61)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(77)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(93)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(109)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(125)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(30)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(46)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(62)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(78)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(94)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(110)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(126)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
    T_batch_matmul_NT_local[(0)] = (T_batch_matmul_NT_local[(0)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
    T_batch_matmul_NT_local[(1)] = (T_batch_matmul_NT_local[(1)] + (placeholder_d_shared[(31)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
    T_batch_matmul_NT_local[(2)] = (T_batch_matmul_NT_local[(2)] + (placeholder_d_shared[(47)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
    T_batch_matmul_NT_local[(3)] = (T_batch_matmul_NT_local[(3)] + (placeholder_d_shared[(63)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
    T_batch_matmul_NT_local[(4)] = (T_batch_matmul_NT_local[(4)] + (placeholder_d_shared[(79)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
    T_batch_matmul_NT_local[(5)] = (T_batch_matmul_NT_local[(5)] + (placeholder_d_shared[(95)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
    T_batch_matmul_NT_local[(6)] = (T_batch_matmul_NT_local[(6)] + (placeholder_d_shared[(111)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
    T_batch_matmul_NT_local[(7)] = (T_batch_matmul_NT_local[(7)] + (placeholder_d_shared[(127)] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    T_batch_matmul_NT[((((((((int)blockIdx.x) >> 6) * 8192) + (i_inner * 1024)) + ((((int)blockIdx.x) & 63) * 16)) + ((int)threadIdx.x)))] = T_batch_matmul_NT_local[(i_inner)];
  }
}

int main(int argc, char const* argv[]) {
  return 0;
}
