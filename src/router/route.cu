#include "hip/hip_runtime.h"

#include <brt/router/route.h>

namespace brt {
namespace router {

__global__ void __launch_bounds__(1024) no_transform_route_with_out_indices(
    float* __restrict__ in_data /*[sample_num x sample_dim]*/,
    float* __restrict__ out_data /*[?load*dst_num x sample_dim]*/,
    int* __restrict__ route_indices /*[sample_num x dst_num]*/,
    int* __restrict__ dst_loads /*[dst_num]*/, int sample_num, int sample_dim, int dst_num) {
  for (int i = blockIdx.x; i < sample_num; i += gridDim.x) {
    int route_index = i * dst_num + blockIdx.y;
    int local_dst = route_indices[route_index];
    if (local_dst == 0) {
      continue;
    }
    int global_dst = local_dst - 1;
    for (int j = 0; j < blockIdx.y; j++) {
      global_dst += dst_loads[j];
    }
    for (int j = threadIdx.x; j < sample_dim; j += 1024) {
      out_data[global_dst * sample_dim + j] = in_data[i * sample_dim + j];
    }
  }
}

__global__ void __launch_bounds__(1024)
    route_with_out_indices(float* __restrict__ in_data /*[sample_num x sample_dim]*/,
                           float* __restrict__ out_data /*[?load*dst_num x sample_dim]*/,
                           float* __restrict__ gates /*[sample_num x dst_num]*/,
                           int* __restrict__ route_indices /*[sample_num x dst_num]*/,
                           int* __restrict__ dst_loads /*[dst_num]*/, int sample_num,
                           int sample_dim, int dst_num) {
  for (int i = blockIdx.x; i < sample_num; i += gridDim.x) {
    int route_index = i * dst_num + blockIdx.y;

    int local_dst = route_indices[route_index];
    if (local_dst == 0) {
      continue;
    }
    int global_dst = local_dst - 1;
    for (int j = 0; j < blockIdx.y; j++) {
      global_dst += dst_loads[j];
    }

    for (int j = threadIdx.x; j < sample_dim; j += 1024) {
      out_data[global_dst * sample_dim + j] = in_data[i * sample_dim + j] * gates[route_index];
    }
  }
}

__global__ void __launch_bounds__(1024) no_transform_route_back_with_in_indices(
    float* __restrict__ in_data /*[?load*dst_num x sample_dim]*/,
    float* __restrict__ out_data /*[sample_num x sample_dim]*/,
    int* __restrict__ route_indices /*[sample_num x dst_num]*/,
    int* __restrict__ dst_loads /*[dst_num]*/, int sample_num, int sample_dim, int dst_num) {
  for (int i = blockIdx.x; i < sample_num; i += gridDim.x) {
    for (int j = 0; j < dst_num; j++) {
      int route_index = i * dst_num + j;
      int local_dst = route_indices[route_index];
      if (local_dst == 0) {
        continue;
      }
      int global_dst = local_dst - 1;
      for (int k = 0; k < j; k++) {
        global_dst += dst_loads[k];
      }
      for (int k = threadIdx.x; k < sample_dim; k += 1024) {
        out_data[i * sample_dim + k] += in_data[global_dst * sample_dim + k];
      }
    }
  }
}

__global__ void __launch_bounds__(1024)
    route_back_with_in_indices(float* __restrict__ in_data /*[?load*dst_num x sample_dim]*/,
                               float* __restrict__ out_data /*[sample_num x sample_dim]*/,
                               float* __restrict__ gates /*[sample_num x dst_num]*/,
                               int* __restrict__ route_indices /*[sample_num x dst_num]*/,
                               int* __restrict__ dst_loads /*[dst_num]*/, int sample_num,
                               int sample_dim, int dst_num) {
  for (int i = blockIdx.x; i < sample_num; i += gridDim.x) {
    for (int j = 0; j < dst_num; j++) {
      int route_index = i * dst_num + j;
      int local_dst = route_indices[route_index];
      if (local_dst == 0) {
        continue;
      }
      int global_dst = local_dst - 1;
      for (int k = 0; k < j; k++) {
        global_dst += dst_loads[k];
      }
      for (int k = threadIdx.x; k < sample_dim; k += 1024) {
        out_data[i * sample_dim + k] += in_data[global_dst * sample_dim + k] * gates[route_index];
      }
    }
  }
}

__global__ void __launch_bounds__(1024) no_transform_route_with_in_indices(
    float* __restrict__ in_data /*[sample_num x sample_dim]*/,
    float* __restrict__ out_data /*[?load*path_num x sample_dim]*/,
    int* __restrict__ route_indices /*[sample_num x path_num]*/,
    int* __restrict__ path_loads /*[path_num]*/, int* __restrict__ capacities /*[path_num] */,
    int sample_dim, int path_num) {
  // [thread_extent] blockIdx.x = capacities max
  // [thread_extent] blockIdx.y = 4
  // [thread_extent] threadIdx.x = 1024
  for (int path_idx = blockIdx.y; path_idx < path_num; path_idx += gridDim.y) {
    int base_index = 0;
    for (int i = 0; i < path_idx; i++) {
      base_index += capacities[i];
    }
    for (int sample_idx = blockIdx.x;
         sample_idx < path_loads[path_idx] && sample_idx < capacities[path_idx];
         sample_idx += gridDim.x) {
      int global_idx = sample_idx * path_num + path_idx;
      int out_data_index = (base_index + sample_idx) * sample_dim;
      int in_data_index = route_indices[global_idx] * sample_dim;

      for (int j = threadIdx.x; j < sample_dim; j += 1024) {
        out_data[out_data_index + j] = in_data[in_data_index + j];
      }
    }
  }
}

__global__ void __launch_bounds__(1024)
    route_with_in_indices(float* __restrict__ in_data /*[sample_num x sample_dim]*/,
                          float* __restrict__ out_data /*[?load*path_num x sample_dim]*/,
                          float* __restrict__ gates /*[sample_num x dst_num]*/,
                          int* __restrict__ route_indices /*[sample_num x path_num]*/,
                          int* __restrict__ path_loads /*[path_num]*/,
                          int* __restrict__ capacities /*[path_num] */, int sample_dim,
                          int path_num) {
  // [thread_extent] blockIdx.x = 128
  // [thread_extent] blockIdx.y = 4
  // [thread_extent] threadIdx.x = 1024

  for (int path_idx = blockIdx.y; path_idx < path_num; path_idx += gridDim.y) {
    int base_index = 0;
    for (int i = 0; i < path_idx; i++) {
      base_index += capacities[i];
    }
    for (int sample_idx = blockIdx.x;
         sample_idx < path_loads[path_idx] && sample_idx < capacities[path_idx];
         sample_idx += gridDim.x) {
      int global_idx = sample_idx * path_num + path_idx;
      int out_data_index = (base_index + sample_idx) * sample_dim;
      int in_data_index = route_indices[global_idx] * sample_dim;

      for (int j = threadIdx.x; j < sample_dim; j += 1024) {
        out_data[out_data_index + j] = in_data[in_data_index + j] * gates[global_idx];
      }
    }
  }
}

__global__ void __launch_bounds__(1024) atomic_no_transform_route_back_with_out_indices(
    float* __restrict__ in_data /*[sample_num x sample_dim]*/,
    float* __restrict__ out_data /*[?load*path_num x sample_dim]*/,
    int* __restrict__ route_indices /*[sample_num x path_num]*/,
    int* __restrict__ path_loads /*[path_num]*/, int* __restrict__ capacities /*[path_num] */,
    int sample_dim, int path_num) {
  // [thread_extent] blockIdx.x = 128
  // [thread_extent] blockIdx.y = 4
  // [thread_extent] threadIdx.x = 1024

  for (int path_idx = blockIdx.y; path_idx < path_num; path_idx += gridDim.y) {
    int base_index = 0;
    for (int i = 0; i < path_idx; i++) {
      base_index += capacities[i];
    }
    for (int sample_idx = blockIdx.x;
         sample_idx < path_loads[path_idx] && sample_idx < capacities[path_idx];
         sample_idx += gridDim.x) {
      int global_idx = sample_idx * path_num + path_idx;
      int in_data_index = (base_index + sample_idx) * sample_dim;
      int out_data_index = route_indices[global_idx] * sample_dim;

      for (int j = threadIdx.x; j < sample_dim; j += 1024) {
        atomicAdd(&out_data[out_data_index + j], in_data[in_data_index + j]);
      }
    }
  }
}

__global__ void __launch_bounds__(1024) no_transform_route_back_with_out_indices(
    float* __restrict__ in_data /*[?load*path_num x sample_dim]*/,
    float* __restrict__ out_data /*[sample_num x sample_dim]*/,
    int* __restrict__ route_indices /*[sample_num x path_num]*/,
    int* __restrict__ path_loads /*[path_num]*/, int* __restrict__ capacities /*[path_num] */,
    int sample_dim, int path_num) {
  // [thread_extent] blockIdx.x = 512
  // [thread_extent] threadIdx.x = 1024

  int base_index = 0;
  for (int path_idx = 0; path_idx < path_num; path_idx++) {
    for (int sample_idx = blockIdx.x;
         sample_idx < path_loads[path_idx] && sample_idx < capacities[path_idx];
         sample_idx += gridDim.x) {
      int global_idx = sample_idx * path_num + path_idx;
      int in_data_index = (base_index + sample_idx) * sample_dim;
      int out_data_index = route_indices[global_idx] * sample_dim;

      for (int j = threadIdx.x; j < sample_dim; j += 1024) {
        out_data[out_data_index + j] += in_data[in_data_index + j];
      }
    }
    base_index += capacities[path_idx];
  }
}

__global__ void __launch_bounds__(1024) atomic_route_back_with_out_indices(
    float* __restrict__ in_data /*[?load*path_num x sample_dim]*/,
    float* __restrict__ out_data /*[sample_num x sample_dim]*/,
    float* __restrict__ gates /*[sample_num x dst_num]*/,
    int* __restrict__ route_indices /*[sample_num x path_num]*/,
    int* __restrict__ path_loads /*[path_num]*/, int* __restrict__ capacities /*[path_num] */,
    int sample_dim, int path_num) {
  // [thread_extent] blockIdx.x = 128
  // [thread_extent] blockIdx.y = 4
  // [thread_extent] threadIdx.x = 1024

  for (int path_idx = blockIdx.y; path_idx < path_num; path_idx += gridDim.y) {
    int base_index = 0;
    for (int i = 0; i < path_idx; i++) {
      base_index += capacities[i];
    }
    for (int sample_idx = blockIdx.x;
         sample_idx < path_loads[path_idx] && sample_idx < capacities[path_idx];
         sample_idx += gridDim.x) {
      int global_idx = sample_idx * path_num + path_idx;
      int in_data_index = (base_index + sample_idx) * sample_dim;
      int out_data_index = route_indices[global_idx] * sample_dim;

      for (int j = threadIdx.x; j < sample_dim; j += 1024) {
        atomicAdd(&out_data[out_data_index + j], in_data[in_data_index + j] * gates[global_idx]);
      }
    }
  }
}

__global__ void __launch_bounds__(1024)
    route_back_with_out_indices(float* __restrict__ in_data /*[?load*path_num x sample_dim]*/,
                                float* __restrict__ out_data /*[sample_num x sample_dim]*/,
                                float* __restrict__ gates /*[sample_num x dst_num]*/,
                                int* __restrict__ route_indices /*[sample_num x path_num]*/,
                                int* __restrict__ path_loads /*[path_num]*/,
                                int* __restrict__ capacities /*[path_num] */, int sample_dim,
                                int path_num) {
  // [thread_extent] blockIdx.x = 512
  // [thread_extent] threadIdx.x = 1024

  int base_index = 0;
  for (int path_idx = 0; path_idx < path_num; path_idx++) {
    for (int sample_idx = blockIdx.x;
         sample_idx < path_loads[path_idx] && sample_idx < capacities[path_idx];
         sample_idx += gridDim.x) {
      int global_idx = sample_idx * path_num + path_idx;
      int in_data_index = (base_index + sample_idx) * sample_dim;
      int out_data_index = route_indices[global_idx] * sample_dim;

      for (int j = threadIdx.x; j < sample_dim; j += 1024) {
        out_data[out_data_index + j] += in_data[in_data_index + j] * gates[global_idx];
      }
    }
    base_index += capacities[path_idx];
  }
}

void RouteWithInDataIndices(float* in_data /*[sample_num x sample_dim]*/,
                            float* outdata /*[?load*dst_num x sample_dim]*/,
                            float* gates /*[sample_num x dst_num]*/,
                            int* route_indices /*[sample_num x dst_num]*/,
                            int* path_loads /*[dst_num]*/, int* capacities /*[dst_num]*/,
                            int sample_dim, int dst_num, hipStream_t stream) {
  constexpr dim3 block_size(1024);
  constexpr dim3 grid_size(128, 4);
  if (gates == nullptr) {
    no_transform_route_with_in_indices<<<grid_size, block_size, 0, stream>>>(
        in_data, outdata, route_indices, path_loads, capacities, sample_dim, dst_num);
  } else {
    route_with_in_indices<<<grid_size, block_size, 0, stream>>>(
        in_data, outdata, gates, route_indices, path_loads, capacities, sample_dim, dst_num);
  }
}

void RouteBackWithOutDataIndices(float* in_data /*[?load*dst_num x sample_dim]*/,
                                 float* outdata /*[sample_num x sample_dim]*/,
                                 float* gates /*[sample_num x dst_num]*/,
                                 int* route_indices /*[sample_num x dst_num]*/,
                                 int* dst_loads /*[dst_num]*/, int sample_num, int sample_dim,
                                 int dst_num, hipStream_t stream) {
  constexpr dim3 block_size(1024);
  dim3 grid_size(512);
  if (gates == nullptr) {
    no_transform_route_back_with_in_indices<<<grid_size, block_size, 0, stream>>>(
        in_data, outdata, route_indices, dst_loads, sample_num, sample_dim, dst_num);
  } else {
    route_back_with_in_indices<<<grid_size, block_size, 0, stream>>>(
        in_data, outdata, gates, route_indices, dst_loads, sample_num, sample_dim, dst_num);
  }
}

void RouteWithLocalIndices(float* in_data /*[sample_num x sample_dim]*/,
                           float* outdata /*[?load*dst_num x sample_dim]*/,
                           float* gates /*[sample_num x dst_num]*/,
                           int* route_indices /*[sample_num x dst_num]*/,
                           int* dst_loads /*[dst_num]*/, int sample_num, int sample_dim,
                           int dst_num, hipStream_t stream) {
  constexpr dim3 block_size(1024);
  dim3 grid_size(512, dst_num);
  if (gates == nullptr) {
    no_transform_route_with_out_indices<<<grid_size, block_size, 0, stream>>>(
        in_data, outdata, route_indices, dst_loads, sample_num, sample_dim, dst_num);
  } else {
    route_with_out_indices<<<grid_size, block_size, 0, stream>>>(
        in_data, outdata, gates, route_indices, dst_loads, sample_num, sample_dim, dst_num);
  }
}

void RouteBackWithLocalIndices(float* in_data /*[?load*dst_num x sample_dim]*/,
                               float* outdata /*[sample_num x sample_dim]*/,
                               float* gates /*[sample_num x dst_num]*/,
                               int* route_indices /*[sample_num x dst_num]*/,
                               int* dst_loads /*[dst_num]*/, int sample_num, int sample_dim,
                               int dst_num, hipStream_t stream) {
  constexpr dim3 block_size(1024);
  dim3 grid_size(512);
  if (gates == nullptr) {
    no_transform_route_back_with_in_indices<<<grid_size, block_size, 0, stream>>>(
        in_data, outdata, route_indices, dst_loads, sample_num, sample_dim, dst_num);
  } else {
    route_back_with_in_indices<<<grid_size, block_size, 0, stream>>>(
        in_data, outdata, gates, route_indices, dst_loads, sample_num, sample_dim, dst_num);
  }
}

}  // namespace router
}  // namespace brt