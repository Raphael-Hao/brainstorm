#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2022 by Microsoft Corporation.
 * Licensed under the MIT license.
 */

#include <brt/jit/compiler.h>
#include <brt/runtime/cuda_utils.h>
#include <dmlc/common.h>

#include "./ptr_arith.cuh"
#include "./utils.h"

namespace brt {
namespace jit {

void KernelConfig::InitBranchArgStore() {
  this->standalone_arg_num = this->arg_num - this->shared_arg_num;
  CHECK_GT(this->branch_num, 0);
  CHECK_GT(this->supported_capacity_num, 0);
  CHECK_GT(this->arg_num, 0);
  CHECK_EQ(this->arg_num, this->shared_arg_num + this->standalone_arg_num);
  CUDA_CHECK(hipHostMalloc(&this->shared_arg_offset, sizeof(int) * this->branch_num));
  this->standalone_arg_hptr_array.resize(this->standalone_arg_num, nullptr);
  for (auto& host_ptr : this->standalone_arg_hptr_array) {
    CUDA_CHECK(hipHostMalloc(&host_ptr, sizeof(void*) * this->branch_num));
  }
  this->arg_dptr_array.resize(this->arg_num, nullptr);
  for (auto& device_ptr : this->arg_dptr_array) {
    CUDA_CHECK(hipMalloc(&device_ptr, sizeof(void*) * this->branch_num));
  }
  for (auto i = 0; i < this->supported_capacity_num; i++) {
    this->capacity_index_map[this->supported_capacities[i]] = i;
  }
}

CUDACompiler::CUDACompiler() {}

CUDACompiler::~CUDACompiler() {}

CUDACompiler& CUDACompiler::GetCompiler() {
  static CUDACompiler instance;
  return instance;
}

std::string CUDACompiler::RTCompile(const char* code, const std::string& arch) {
  std::string arch_option = "--gpu-architecture=compute_" + arch;
  std::vector<const char*> param_cstrings = {
      "--restrict",        "--include-path=/usr/local/cuda/include",
      arch_option.c_str(), "--use_fast_math",
      "--std=c++14",       "--extra-device-vectorization"};
  hiprtcProgram prog;
  NVRTC_CHECK(hiprtcCreateProgram(&prog, code, nullptr, 0, nullptr, nullptr));
  hiprtcResult nvrtc_compile_result =
      hiprtcCompileProgram(prog, param_cstrings.size(), param_cstrings.data());

  if (nvrtc_compile_result != HIPRTC_SUCCESS) {
    size_t log_size;
    NVRTC_CHECK(hiprtcGetProgramLogSize(prog, &log_size));
    std::string log;
    log.resize(log_size);
    NVRTC_CHECK(hiprtcGetProgramLog(prog, &log[0]));
    LOG(FATAL) << "hiprtcCompileProgram failed: \n" << log;
  }

  size_t ptx_size;
  NVRTC_CHECK(hiprtcGetCodeSize(prog, &ptx_size));

  std::string ptx;
  ptx.resize(ptx_size);
  NVRTC_CHECK(hiprtcGetCode(prog, &ptx[0]));
  NVRTC_CHECK(hiprtcDestroyProgram(&prog));
  return ptx;
}

hipFunction_t CUDACompiler::Activate(int fd, int dev) {
  auto& kernel = kernels_[fd];
  if (kernel.hFunc.size() <= static_cast<size_t>(dev)) kernel.hFunc.resize(dev + 1);

  if (kernel.hFunc[dev] == nullptr) {
    int major, minor;
    CUDA_CHECK(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, dev));
    CUDA_CHECK(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, dev));
    std::string arch = std::to_string(major) + std::to_string(minor);

    const char* source = kernel.code.data();

    std::string image;
    image = RTCompile(source, arch);
    long launch_bound =
        CaptureWithDefault(kernel.code, std::regex(R"(\s+__launch_bounds__\((\d+)\)\s+)"), 0);

    long max_registers = 65536 / launch_bound / 2;
    static hipJitOption options[] = {hipJitOptionOptimizationLevel, hipJitOptionThreadsPerBlock,
                                     hipJitOptionMaxRegisters};
    static void* values[] = {(void*)4L, (void*)launch_bound, (void*)max_registers};

    hipModule_t hMod = nullptr;
    CU_CHECK(hipModuleLoadDataEx(&hMod, image.c_str(), sizeof(options) / sizeof(*options), options,
                                values));
    CHECK(nullptr != hMod);

    int func_entry = image.find(" .entry ");
    func_entry += 8;
    int func_end = image.find("(", func_entry);
    std::string func_name = image.substr(func_entry, func_end - func_entry);
    kernel.fname = func_name;
    CU_CHECK(hipModuleGetFunction(&kernel.hFunc[dev], hMod, func_name.c_str()));
    CHECK(nullptr != kernel.hFunc[dev]);
  }

  return kernel.hFunc[dev];
}

void CUDACompiler::Execute(const std::vector<const void*>& ppargs, int fd, int dev,
                           hipStream_t stream) {
  hipFunction_t hfunc = Activate(fd, dev);
  auto& blocks = kernels_[fd].blocks;
  auto& threads = kernels_[fd].threads;
  CHECK_EQ(0, hipModuleLaunchKernel(hfunc, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z,
                             0, stream, (void**)ppargs.data(), nullptr));
}

void CUDACompiler::StaticExecute(const std::vector<const void*>& ppargs, int fd, int dev,
                                 hipStream_t stream) {
  hipFunction_t hfunc = Activate(fd, dev);
  auto& blocks = kernels_[fd].blocks;
  auto& threads = kernels_[fd].threads;

  CHECK_EQ(0, hipModuleLaunchKernel(hfunc, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z,
                             0, stream, (void**)ppargs.data(), nullptr));
}

void CUDACompiler::HeteroExecute(const std::vector<const void*>& ppargs,
                                 const std::vector<long>& active_blocks, int fd, int dev,
                                 hipStream_t stream) {
  hipFunction_t hfunc = Activate(fd, dev);
  auto& blocks = kernels_[fd].blocks;
  auto& threads = kernels_[fd].threads;
  CHECK_EQ(kernels_[fd].grid_sizes.size(), active_blocks.size());
  blocks.x = 0;
  threads.x = 0;
  for (size_t i = 0; i < active_blocks.size(); ++i) {
    if (active_blocks[i] == 0) continue;
    blocks.x += kernels_[fd].grid_sizes[i];
    threads.x = std::max(threads.x, kernels_[fd].block_sizes[i]);
  }
  CHECK_EQ(0, hipModuleLaunchKernel(hfunc, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z,
                             0, stream, (void**)ppargs.data(), nullptr));
}

void CUDACompiler::HomoExecute(const std::vector<const void*>& shared_inputs_ptr,
                               const std::vector<const void*>& standalone_inputs_ptr,
                               const std::vector<int>& branch_capacities, int fd, int dev,
                               hipStream_t stream) {
  auto& kernel = kernels_[fd];

  // for (auto i = 0; i < kernel.shared_arg_num; i++) {
  //   printf("shared_inputs_ptr[%d] = %p\n", i, shared_inputs_ptr[i]);
  // }

  // for (auto i = 0; i < kernel.standalone_arg_num * kernel.branch_num; i++) {
  //   printf("standalone_inputs_ptr[%d] = %p\n", i, standalone_inputs_ptr[i]);
  // }

  std::vector<int> active_blocks(kernel.supported_capacity_num, 0);
  // reorder the arguments for kernel based on capacities
  auto branch_indice_with_order = SortIndice(branch_capacities);

  int real_branch_index = 0;
  // printf("runtime arg dispatch begin\n");
  for (auto branch_idx = 0; branch_idx < kernel.branch_num; branch_idx++) {
    auto& branch_idx_in_order = branch_indice_with_order[branch_idx];
    // printf("sorted branch: %d -> origin branch %d\n", branch_idx, branch_idx_in_order);
    auto& capacity = branch_capacities[branch_idx_in_order];
    // printf("capacity: %d\n", capacity);
    if (capacity == 0) continue;
    active_blocks[kernel.capacity_index_map[capacity]]++;
    // printf("active_blocks[%d] capacity updated to: %d\n", kernel.capacity_index_map[capacity],
    //  active_blocks[kernel.capacity_index_map[capacity]]);
    auto shared_arg_branch_index = std::accumulate(
        branch_capacities.begin(), branch_capacities.begin() + branch_idx_in_order, 0);
    // printf("shared_arg_branch_index: %d for branch: %d, real: %d\n", shared_arg_branch_index,
    //  branch_idx, real_branch_index);
    kernel.shared_arg_offset[real_branch_index] = shared_arg_branch_index;
    // printf("kernel.shared_arg_offset[%d] = %d\n", real_branch_index,
    //  kernel.shared_arg_offset[real_branch_index]);

    for (auto arg_idx = 0; arg_idx < (kernel.arg_num - kernel.shared_arg_num); arg_idx++) {
      kernel.standalone_arg_hptr_array[arg_idx][real_branch_index] =
          (void*)standalone_inputs_ptr[kernel.standalone_arg_num * branch_idx_in_order + arg_idx];
      // printf("branch: %d, standalone_arg_hptr_array[%d][%d] = %p\n", branch_idx, arg_idx,
      //  real_branch_index, kernel.standalone_arg_hptr_array[arg_idx][real_branch_index]);
    }
    real_branch_index++;
  }
  // print debug info
  // printf("runtime arg dispatch end\n");

  for (auto arg_idx = 0; arg_idx < kernel.arg_num; arg_idx++) {
    if (arg_idx < kernel.shared_arg_num) {
      DevicePtr2PtrArray((char**)kernel.arg_dptr_array[arg_idx], (char*)shared_inputs_ptr[arg_idx],
                         kernel.shared_arg_offset, kernel.branch_num,
                         kernel.shared_arg_grans[arg_idx], stream);
      // CUDA_CHECK(hipStreamSynchronize(stream));
    } else {
      CUDA_CHECK(hipMemcpyAsync(kernel.arg_dptr_array[arg_idx],
                                 kernel.standalone_arg_hptr_array[arg_idx - kernel.shared_arg_num],
                                 real_branch_index * sizeof(void*), hipMemcpyHostToDevice,
                                 stream));
      // CUDA_CHECK(hipStreamSynchronize(stream));
    }
  }
  // for (auto cap_idx = 0; cap_idx < kernel.supported_capacity_num; cap_idx++) {
  //   printf("active_blocks[%d] = %d\n", cap_idx, active_blocks[cap_idx]);
  // }
  // geneerate culaunch config
  std::vector<const void*> pargs(kernel.arg_dptr_array.size() + active_blocks.size()),
      ppargs(kernel.arg_dptr_array.size() + active_blocks.size());
  for (int i = 0; i < (int)kernel.arg_num; ++i) {
    pargs[i] = kernel.arg_dptr_array[i];
    ppargs[i] = &pargs[i];
  }
  for (int i = (int)kernel.arg_num; i < (int)pargs.size(); ++i) {
    pargs[i] = (void*)active_blocks[i - kernel.arg_num];
    ppargs[i] = &pargs[i];
    // ppargs[i] = (void*)&active_blocks[i - kernel.arg_num];
  }

  hipFunction_t hfunc = Activate(fd, dev);
  auto& blocks = kernels_[fd].blocks;
  auto& threads = kernels_[fd].threads;
  CHECK_EQ(kernels_[fd].grid_sizes.size(), active_blocks.size());
  blocks.x = 0;
  threads.x = 0;
  for (size_t i = 0; i < active_blocks.size(); ++i) {
    if (active_blocks[i] == 0) continue;
    blocks.x += kernels_[fd].grid_sizes[i] * active_blocks[i];
    threads.x = std::max(threads.x, kernels_[fd].block_sizes[i]);
  }

  // printf("blocks: %d, %d, %d\n", blocks.x, blocks.y, blocks.z);
  // printf("threads: %d, %d, %d\n", threads.x, threads.y, threads.z);

  CHECK_EQ(0, hipModuleLaunchKernel(hfunc, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z,
                             0, stream, (void**)ppargs.data(), nullptr));
}

std::pair<std::string, int> CUDACompiler::InjectSource(const std::string& headless_code) {
  int fd = kernels_.size();
  kernels_.resize(fd + 1);

  auto& kernel = kernels_[fd];
  kernel.code = "#include <hip/hip_runtime.h>\n#include <hip/hip_fp16.h>\n" + headless_code;

  std::string kernel_type_str = CaptureWithDefault(
      headless_code, std::regex(R"(\/\/\s+\[kernel_type\]\s+(\w+)\s*)"), "global");
  auto kernel_type_it = kernel_type_tb.find(kernel_type_str);
  if (kernel_type_it == kernel_type_tb.end()) {
    LOG(FATAL) << "unknown kernel type: " << kernel_type_str;
  } else {
    kernel.type = kernel_type_it->second;
  }

  switch (kernel.type) {
    case KernelType::kGlobal:
    case KernelType::kHorizFuse: {
      kernel.blocks.x = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.x\s*=\s*(\d+)\s*)"), 1);
      kernel.threads.x = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.x\s*=\s*(\d+)\s*)"), 1);
      break;
    }
    case KernelType::kHeteroFuse: {
      auto fused_kernel_grids_str = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.x\s*=\s*\[([0-9,\s]+)\])"),
          "");
      kernel.grid_sizes = ToUintVector(fused_kernel_grids_str, ',');
      auto fused_kernel_blocks_str = CaptureWithDefault(
          kernel.code,
          std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.x\s*=\s*\[([0-9,\s]+)\])"), "");
      kernel.block_sizes = ToUintVector(fused_kernel_blocks_str, ',');
      break;
    }
    case KernelType::kHomoFuse: {
      auto fused_kernel_grids_str = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.x\s*=\s*\[([0-9,\s]+)\])"),
          "");
      kernel.grid_sizes = ToUintVector(fused_kernel_grids_str, ',');
      auto fused_kernel_blocks_str = CaptureWithDefault(
          kernel.code,
          std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.x\s*=\s*\[([0-9,\s]+)\])"), "");
      kernel.block_sizes = ToUintVector(fused_kernel_blocks_str, ',');
      kernel.branch_num = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+branch_num\s*=\s*(\d+)\s*)"), 1);
      auto capacity_str = CaptureWithDefault(
          kernel.code,
          std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+supported_capacity\s*=\s*\[([0-9,\s]+)\])"),
          "");
      kernel.supported_capacities = ToIntVector(capacity_str);
      kernel.supported_capacity_num = kernel.supported_capacities.size();
      kernel.arg_num = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+arg_num\s*=\s*(\d+)\s*)"), 1);
      kernel.shared_arg_num = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+shared_arg_num\s*=\s*(\d+)\s*)"),
          1);
      auto shared_arg_grans_str = CaptureWithDefault(
          kernel.code,
          std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+shared_arg_grans\s*=\s*\[([0-9,\s]+)\])"), "");
      kernel.shared_arg_grans = ToUintVector(shared_arg_grans_str, ',');
      kernel.InitBranchArgStore();
      break;
    }
    case KernelType::kElasticHomoFuse: {
      kernel.blocks.x = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.x\s*=\s*(\d+)\s*)"), 1);
      kernel.threads.x = CaptureWithDefault(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.x\s*=\s*(\d+)\s*)"), 1);
      break;
    }
    default:
      LOG(FATAL) << "unknown kernel type";
      break;
  }
  kernel.blocks.y = CaptureWithDefault(
      kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.y\s+=\s+(\d+)\s*)"), 1);
  kernel.blocks.z = CaptureWithDefault(
      kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.z\s+=\s+(\d+)\s*)"), 1);
  kernel.threads.y = CaptureWithDefault(
      kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.y\s+=\s+(\d+)\s*)"), 1);
  kernel.threads.z = CaptureWithDefault(
      kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.z\s+=\s+(\d+)\s*)"), 1);

  return {kernel_type_str, fd};
}

}  // namespace jit
}  // namespace brt
