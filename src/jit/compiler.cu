#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2022 by Microsoft Corporation.
 * Licensed under the MIT license.
 */

#include <brt/jit/compiler.h>
#include <brt/runtime/cuda_utils.h>
#include <dmlc/common.h>

#include "./ptr_arith.cuh"
#include "./utils.h"

namespace brt {
namespace jit {

void KernelConfig::InitBranchArgStore() {
  this->standalone_arg_num = this->arg_num - this->shared_arg_num;
  CHECK_GT(this->branch_num, 0);
  CHECK_GT(this->supported_capacity_num, 0);
  CHECK_GT(this->arg_num, 0);
  CHECK_EQ(this->arg_num, this->shared_arg_num + this->standalone_arg_num);
  CUDA_CHECK(hipHostMalloc(&this->shared_arg_offset, sizeof(int) * this->branch_num));
  this->standalone_arg_hptr_array.resize(this->standalone_arg_num, nullptr);
  for (auto& host_ptr : this->standalone_arg_hptr_array) {
    CUDA_CHECK(hipHostMalloc(&host_ptr, sizeof(void*) * this->branch_num));
  }
  this->arg_dptr_array.resize(this->arg_num, nullptr);
  for (auto& device_ptr : this->arg_dptr_array) {
    CUDA_CHECK(hipMalloc(&device_ptr, sizeof(void*) * this->branch_num));
  }
  for (auto i = 0; i < this->supported_capacity_num; i++) {
    this->capacity_index_map[this->supported_capacities[i]] = i;
  }
}

CUDACompiler::CUDACompiler() {}

CUDACompiler::~CUDACompiler() {}

CUDACompiler& CUDACompiler::get_compiler() {
  static CUDACompiler instance;
  return instance;
}

std::string CUDACompiler::nvrtc_compile(const char* code, const std::string& arch) {
  std::string arch_option = "--gpu-architecture=compute_" + arch;
  std::vector<const char*> param_cstrings = {
      "--restrict",        "--include-path=/usr/local/cuda/include",
      arch_option.c_str(), "--use_fast_math",
      "--std=c++14",       "--extra-device-vectorization"};
  hiprtcProgram prog;
  NVRTC_CHECK(hiprtcCreateProgram(&prog, code, nullptr, 0, nullptr, nullptr));
  hiprtcResult nvrtc_compile_result =
      hiprtcCompileProgram(prog, param_cstrings.size(), param_cstrings.data());

  if (nvrtc_compile_result != HIPRTC_SUCCESS) {
    size_t log_size;
    NVRTC_CHECK(hiprtcGetProgramLogSize(prog, &log_size));
    std::string log;
    log.resize(log_size);
    NVRTC_CHECK(hiprtcGetProgramLog(prog, &log[0]));
    LOG(FATAL) << "hiprtcCompileProgram failed: \n" << log;
  }

  size_t ptx_size;
  NVRTC_CHECK(hiprtcGetCodeSize(prog, &ptx_size));

  std::string ptx;
  ptx.resize(ptx_size);
  NVRTC_CHECK(hiprtcGetCode(prog, &ptx[0]));
  NVRTC_CHECK(hiprtcDestroyProgram(&prog));
  return ptx;
}

hipFunction_t CUDACompiler::activate(int fd, int dev) {
  auto& kernel = kernels_[fd];
  if (kernel.hFunc.size() <= static_cast<size_t>(dev)) kernel.hFunc.resize(dev + 1);

  if (kernel.hFunc[dev] == nullptr) {
    int major, minor;
    CUDA_CHECK(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, dev));
    CUDA_CHECK(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, dev));
    std::string arch = std::to_string(major) + std::to_string(minor);

    const char* source = kernel.code.data();

    std::string image;
    image = nvrtc_compile(source, arch);
    long launch_bound =
        capture_with_default(kernel.code, std::regex(R"(\s+__launch_bounds__\((\d+)\)\s+)"), 0);

    static hipJitOption options[] = {hipJitOptionOptimizationLevel, hipJitOptionThreadsPerBlock};
    static void* values[] = {(void*)4L, (void*)launch_bound};

    hipModule_t hMod = nullptr;
    CU_CHECK(hipModuleLoadDataEx(&hMod, image.c_str(), sizeof(options) / sizeof(*options), options,
                                values));
    CHECK(nullptr != hMod);

    int func_entry = image.find(" .entry ");
    func_entry += 8;
    int func_end = image.find("(", func_entry);
    std::string func_name = image.substr(func_entry, func_end - func_entry);
    kernel.fname = func_name;
    CU_CHECK(hipModuleGetFunction(&kernel.hFunc[dev], hMod, func_name.c_str()));
    CHECK(nullptr != kernel.hFunc[dev]);
  }

  return kernel.hFunc[dev];
}

void CUDACompiler::execute(const std::vector<const void*>& ppargs, int fd, int dev,
                           hipStream_t stream) {
  hipFunction_t hfunc = activate(fd, dev);
  auto& blocks = kernels_[fd].blocks;
  auto& threads = kernels_[fd].threads;
  CHECK_EQ(0, hipModuleLaunchKernel(hfunc, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z,
                             0, stream, (void**)ppargs.data(), nullptr));
}

void CUDACompiler::static_execute(const std::vector<const void*>& ppargs, int fd, int dev,
                                  hipStream_t stream) {
  hipFunction_t hfunc = activate(fd, dev);
  auto& blocks = kernels_[fd].blocks;
  auto& threads = kernels_[fd].threads;

  CHECK_EQ(0, hipModuleLaunchKernel(hfunc, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z,
                             0, stream, (void**)ppargs.data(), nullptr));
}

void CUDACompiler::hetero_execute(const std::vector<const void*>& ppargs,
                                  const std::vector<long>& active_blocks, int fd, int dev,
                                  hipStream_t stream) {
  hipFunction_t hfunc = activate(fd, dev);
  auto& blocks = kernels_[fd].blocks;
  auto& threads = kernels_[fd].threads;
  CHECK_EQ(kernels_[fd].grid_sizes.size(), active_blocks.size());
  blocks.x = 0;
  threads.x = 0;
  for (size_t i = 0; i < active_blocks.size(); ++i) {
    if (active_blocks[i] == 0) continue;
    blocks.x += kernels_[fd].grid_sizes[i];
    threads.x = std::max(threads.x, kernels_[fd].block_sizes[i]);
  }
  CHECK_EQ(0, hipModuleLaunchKernel(hfunc, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z,
                             0, stream, (void**)ppargs.data(), nullptr));
}

void CUDACompiler::homo_execute(const std::vector<const void*>& shared_inputs_ptr,
                                const std::vector<const void*>& standalone_inputs_ptr,
                                const std::vector<long>& branch_capacities, int fd, int dev,
                                hipStream_t stream) {
  auto& kernel = kernels_[fd];

  // for (auto i = 0; i < kernel.shared_arg_num; i++) {
  //   printf("shared_inputs_ptr[%d] = %p\n", i, shared_inputs_ptr[i]);
  // }

  // for (auto i = 0; i < kernel.standalone_arg_num * kernel.branch_num; i++) {
  //   printf("standalone_inputs_ptr[%d] = %p\n", i, standalone_inputs_ptr[i]);
  // }

  std::vector<int> active_blocks(kernel.supported_capacity_num, 0);
  // reorder the arguments for kernel based on capacities
  auto branch_indice_with_order = sort_indice(branch_capacities);

  int real_branch_index = 0;
  // printf("runtime arg dispatch begin\n");
  for (auto branch_idx = 0; branch_idx < kernel.branch_num; branch_idx++) {
    auto& branch_idx_in_order = branch_indice_with_order[branch_idx];
    // printf("sorted branch: %d -> origin branch %d\n", branch_idx, branch_idx_in_order);
    auto& capacity = branch_capacities[branch_idx_in_order];
    // printf("capacity: %d\n", capacity);
    if (capacity == 0) continue;
    active_blocks[kernel.capacity_index_map[capacity]]++;
    // printf("active_blocks[%d] capacity updated to: %d\n", kernel.capacity_index_map[capacity],
    //  active_blocks[kernel.capacity_index_map[capacity]]);
    auto shared_arg_branch_index = std::accumulate(
        branch_capacities.begin(), branch_capacities.begin() + branch_idx_in_order, 0);
    // printf("shared_arg_branch_index: %d for branch: %d, real: %d\n", shared_arg_branch_index,
    //  branch_idx, real_branch_index);
    kernel.shared_arg_offset[real_branch_index] = shared_arg_branch_index;
    // printf("kernel.shared_arg_offset[%d] = %d\n", real_branch_index,
    //  kernel.shared_arg_offset[real_branch_index]);

    for (auto arg_idx = 0; arg_idx < (kernel.arg_num - kernel.shared_arg_num); arg_idx++) {
      kernel.standalone_arg_hptr_array[arg_idx][real_branch_index] =
          (void*)standalone_inputs_ptr[kernel.standalone_arg_num * branch_idx_in_order + arg_idx];
      // printf("branch: %d, standalone_arg_hptr_array[%d][%d] = %p\n", branch_idx, arg_idx,
      //  real_branch_index, kernel.standalone_arg_hptr_array[arg_idx][real_branch_index]);
    }
    real_branch_index++;
  }
  // print debug info
  // printf("runtime arg dispatch end\n");

  for (auto arg_idx = 0; arg_idx < kernel.arg_num; arg_idx++) {
    if (arg_idx < kernel.shared_arg_num) {
      DevicePtr2PtrArray((char**)kernel.arg_dptr_array[arg_idx], (char*)shared_inputs_ptr[arg_idx],
                         kernel.shared_arg_offset, kernel.branch_num,
                         kernel.shared_arg_grans[arg_idx], stream);
      // CUDA_CHECK(hipStreamSynchronize(stream));
    } else {
      CUDA_CHECK(hipMemcpyAsync(kernel.arg_dptr_array[arg_idx],
                                 kernel.standalone_arg_hptr_array[arg_idx - kernel.shared_arg_num],
                                 real_branch_index * sizeof(void*), hipMemcpyHostToDevice,
                                 stream));
      // CUDA_CHECK(hipStreamSynchronize(stream));
    }
  }
  // for (auto cap_idx = 0; cap_idx < kernel.supported_capacity_num; cap_idx++) {
  //   printf("active_blocks[%d] = %d\n", cap_idx, active_blocks[cap_idx]);
  // }
  // geneerate culaunch config
  std::vector<const void*> pargs(kernel.arg_dptr_array.size() + active_blocks.size()),
      ppargs(kernel.arg_dptr_array.size() + active_blocks.size());
  for (int i = 0; i < (int)kernel.arg_num; ++i) {
    pargs[i] = kernel.arg_dptr_array[i];
    ppargs[i] = &pargs[i];
  }
  for (int i = (int)kernel.arg_num; i < (int)pargs.size(); ++i) {
    pargs[i] = (void*)active_blocks[i - kernel.arg_num];
    ppargs[i] = &pargs[i];
  }

  hipFunction_t hfunc = activate(fd, dev);
  auto& blocks = kernels_[fd].blocks;
  auto& threads = kernels_[fd].threads;
  CHECK_EQ(kernels_[fd].grid_sizes.size(), active_blocks.size());
  blocks.x = 0;
  threads.x = 0;
  for (size_t i = 0; i < active_blocks.size(); ++i) {
    if (active_blocks[i] == 0) continue;
    blocks.x += kernels_[fd].grid_sizes[i] * active_blocks[i];
    threads.x = std::max(threads.x, kernels_[fd].block_sizes[i]);
  }

  // printf("blocks: %d, %d, %d\n", blocks.x, blocks.y, blocks.z);
  // printf("threads: %d, %d, %d\n", threads.x, threads.y, threads.z);

  CHECK_EQ(0, hipModuleLaunchKernel(hfunc, blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z,
                             0, stream, (void**)ppargs.data(), nullptr));
}

std::pair<std::string, int> CUDACompiler::inject_source(const std::string& headless_code) {
  int fd = kernels_.size();
  kernels_.resize(fd + 1);

  auto& kernel = kernels_[fd];
  kernel.code = "#include <hip/hip_runtime.h>\n#include <hip/hip_fp16.h>\n" + headless_code;

  std::string kernel_type_str = capture_with_default(
      headless_code, std::regex(R"(\/\/\s+\[kernel_type\]\s+(\w+)\s*)"), "global");
  auto kernel_type_it = kernel_type_tb.find(kernel_type_str);
  if (kernel_type_it == kernel_type_tb.end()) {
    LOG(FATAL) << "unknown kernel type: " << kernel_type_str;
  } else {
    kernel.type = kernel_type_it->second;
  }

  switch (kernel.type) {
    case KernelType::kGlobal:
    case KernelType::kHorizFuse: {
      kernel.blocks.x = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.x\s*=\s*(\d+)\s*)"), 1);
      kernel.threads.x = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.x\s*=\s*(\d+)\s*)"), 1);
      break;
    }
    case KernelType::kHeteroFuse: {
      auto fused_kernel_grids_str = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.x\s*=\s*\[([0-9,\s]+)\])"),
          "");
      kernel.grid_sizes = to_uint_vector(fused_kernel_grids_str, ',');
      auto fused_kernel_blocks_str = capture_with_default(
          kernel.code,
          std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.x\s*=\s*\[([0-9,\s]+)\])"), "");
      kernel.block_sizes = to_uint_vector(fused_kernel_blocks_str, ',');
      break;
    }
    case KernelType::kHomoFuse: {
      auto fused_kernel_grids_str = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.x\s*=\s*\[([0-9,\s]+)\])"),
          "");
      kernel.grid_sizes = to_uint_vector(fused_kernel_grids_str, ',');
      auto fused_kernel_blocks_str = capture_with_default(
          kernel.code,
          std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.x\s*=\s*\[([0-9,\s]+)\])"), "");
      kernel.block_sizes = to_uint_vector(fused_kernel_blocks_str, ',');
      kernel.branch_num = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+branch_num\s*=\s*(\d+)\s*)"), 1);
      auto capacity_str = capture_with_default(
          kernel.code,
          std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+supported_capacity\s*=\s*\[([0-9,\s]+)\])"),
          "");
      kernel.supported_capacities = to_int_vector(capacity_str);
      kernel.supported_capacity_num = kernel.supported_capacities.size();
      kernel.arg_num = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+arg_num\s*=\s*(\d+)\s*)"), 1);
      kernel.shared_arg_num = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+shared_arg_num\s*=\s*(\d+)\s*)"),
          1);
      auto shared_arg_grans_str = capture_with_default(
          kernel.code,
          std::regex(R"(\/\/\s+\[homo_fuse_info\]\s+shared_arg_grans\s*=\s*\[([0-9,\s]+)\])"), "");
      kernel.shared_arg_grans = to_uint_vector(shared_arg_grans_str, ',');
      kernel.InitBranchArgStore();
      break;
    }
    case KernelType::kElasticHomoFuse: {
      kernel.blocks.x = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.x\s*=\s*(\d+)\s*)"), 1);
      kernel.threads.x = capture_with_default(
          kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.x\s*=\s*(\d+)\s*)"), 1);
      break;
    }
    default:
      LOG(FATAL) << "unknown kernel type";
      break;
  }
  kernel.blocks.y = capture_with_default(
      kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.y\s+=\s+(\d+)\s*)"), 1);
  kernel.blocks.z = capture_with_default(
      kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+blockIdx.z\s+=\s+(\d+)\s*)"), 1);
  kernel.threads.y = capture_with_default(
      kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.y\s+=\s+(\d+)\s*)"), 1);
  kernel.threads.z = capture_with_default(
      kernel.code, std::regex(R"(\/\/\s+\[thread_extent\]\s+threadIdx.z\s+=\s+(\d+)\s*)"), 1);

  return {kernel_type_str, fd};
}

}  // namespace jit
}  // namespace brt
